#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil_inline.h>

#include "polybenchUtilFuncts.h"
// includes, kernels
#include <scan.cu>  // defines prescanArray()

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// regression test functionality
extern "C" 
unsigned int compare( const float* reference, const float* data, 
                     const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
   createOutputFiles("SLA");  
  // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

#ifndef __DEVICE_EMULATION__

    /////haonan
    //unsigned int num_test_iterations = 100;
    //unsigned int num_elements = 1000000; // can support large, non-power-of-2 arrays!
	
	unsigned int num_test_iterations = 1;////////////////////////////////////////////////////////////////////////////
    unsigned int num_elements = 10000; // can support large, non-power-of-2 arrays!///////////////////////////////////////////////////
	//////haonan
#else
    unsigned int num_test_iterations = 1;
    unsigned int num_elements = 10000; // can support large, non-power-of-2 arrays!
#endif
    
    cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);
    cutGetCmdLineArgumenti( argc, (const char**) argv, "i", (int*)&num_test_iterations);
    printf("num_elements %d \n", num_elements);
    unsigned int mem_size = sizeof( float) * num_elements;
    
    unsigned int timerGPU, timerCPU;
    cutilCheckError(cutCreateTimer(&timerCPU));
    cutilCheckError(cutCreateTimer(&timerGPU));

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( mem_size);
      
    // initialize the input data on the host
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
	    //////haonan
        //h_data[i] = 1.0f;//(int)(10 * rand()/32768.f);///////////////////////////////////////////////////////////////////
		
		//h_data[i] = floor( (i) / 16 );
		h_data[i] = i;
		//////haonan
    }

    // compute reference solution
    float* reference = (float*) malloc( mem_size); 
    cutStartTimer(timerCPU);
    for (unsigned int i = 0; i < num_test_iterations; i++)
    {
        computeGold( reference, h_data, num_elements);
    }
    cutStopTimer(timerCPU);

    // allocate device memory input and output arrays
    float* d_idata = NULL;
    float* d_odata = NULL;

    cutilSafeCall( hipMalloc( (void**) &d_idata, mem_size));
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));
    
    // copy host memory to device input array
    cutilSafeCall( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );
    // initialize all the other device arrays to be safe
    cutilSafeCall( hipMemcpy( d_odata, h_data, mem_size, hipMemcpyHostToDevice) );

    printf("Running parallel prefix sum (prescan) of %d elements\n", num_elements);
    printf("This version is work efficient (O(n) adds)\n");
    printf("and has very few shared memory bank conflicts\n\n");

    preallocBlockSums(num_elements);

    /////haonan
    // run once to remove startup overhead
    //prescanArray(d_odata, d_idata, num_elements);
    /////haonan
	
    // Run the prescan
    cutStartTimer(timerGPU);
    for (unsigned int i = 0; i < num_test_iterations; i++)
    {
        //printf("prescanArray\n");
        prescanArray(d_odata, d_idata, num_elements);
    }
    cutStopTimer(timerGPU);


    deallocBlockSums();    

    // copy result from device to host
    cutilSafeCall(hipMemcpy( h_data, d_odata, sizeof(float) * num_elements, 
                               hipMemcpyDeviceToHost));

    // If this is a regression test write the results to a file
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test 
        cutWriteFilef( "./data/result.dat", h_data, num_elements, 0.0);
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        unsigned int result_regtest = cutComparef( reference, h_data, num_elements);
        printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
        printf( "Average GPU execution time: %f ms\n", cutGetTimerValue(timerGPU) / num_test_iterations);
        printf( "CPU execution time:         %f ms\n", cutGetTimerValue(timerCPU) / num_test_iterations);
	
	#define MIN_EPSILON_ERROR1 1e-3f

	 double nIncluded2=0, sumRelativeError_noskip = 0, sumRelativeError_skipzero = 0;
        double nIncluded=0, nDifferent =0, sumDifferent=0, sumRelativeError=0;
	   fprintf(CPUoutputFile, "1\n");
	   fprintf(GPUoutputFile, "1\n");

	for(int i = 0; i < num_elements; ++i) {
	  fprintf(CPUoutputFile, "%f ",reference[i]);
	  fprintf(GPUoutputFile, "%f ", h_data[i]);
	          

	 nDifferent += (fabs(reference[i] - h_data[i])> MIN_EPSILON_ERROR1);
             if (fabs(reference[i] - h_data[i]) > MIN_EPSILON_ERROR)
                   nDifferent++;
              bool include =true;
            double temp = percentDiffOurs(reference[i] , h_data[i],include);
                if(include) {
                   sumRelativeError += temp;
                   nIncluded++;
                }
                bool include2 =true;
                double temp2 = percentDiff_skipzero(reference[i] , h_data[i], include2);
                if(include2) {
                   sumRelativeError_skipzero += temp2;
                   nIncluded2++;
                }

                double temp3 = percentDiff_noskip(reference[i] , h_data[i]);
                sumRelativeError_noskip += temp3;

	}
	       fprintf(CPUoutputFile, "\n");
	          fprintf(GPUoutputFile, "\n");

	double finalPercError = nDifferent *100.0/num_elements;
	printf("ACT_percLossInQoR %E\n", finalPercError);
        printf("ACT_percSumDifferent %E\n", (sumDifferent*100.0)/num_elements);
        printf("ACT_percRelativeError %E\n", (sumRelativeError*100.0)/nIncluded);
        printf("ACT_percRelativeError_skipzero %E\n", (sumRelativeError_skipzero*100.0)/nIncluded2);
        printf("ACT_percRelativeError_noskip %E\n", (sumRelativeError_noskip*100.0)/num_elements);
    }


    printf("\nCheck out the CUDA Data Parallel Primitives Library for more on scan.\n");
    printf("http://www.gpgpu.org/developer/cudpp\n");

    // cleanup memory
    cutDeleteTimer(timerCPU);
    cutDeleteTimer(timerGPU);
    free( h_data);
    free( reference);
    hipFree( d_odata);
    hipFree( d_idata);

    hipDeviceReset();
}
