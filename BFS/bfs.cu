#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAX_THREADS_PER_BLOCK 256

int no_of_nodes;
int edge_list_size;
FILE *fp;

// Structure to hold a node information
struct Node
{
    int starting;
    int no_of_edges;
};

// Kernel function prototype
__global__ void Kernel(Node* d_graph_nodes, int* d_graph_edges, bool* d_graph_mask, bool* d_graph_visited, int* d_cost, bool* d_over, int no_of_nodes);

// Function declaration
void BFSGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    no_of_nodes = 0;
    edge_list_size = 0;
    BFSGraph(argc, argv);
    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
void BFSGraph(int argc, char** argv) 
{
    printf("Reading File\n");
    static char *input_file_name;

    if (argc == 2) {
        input_file_name = argv[1];
        printf("Input file: %s\n", input_file_name);
    }
    else 
    {
        input_file_name = "SampleGraph.txt";
        printf("No input file specified, defaulting to SampleGraph.txt\n");
    }
    
    // Read in Graph from a file
    fp = fopen(input_file_name, "r");
    if (!fp)
    {
        printf("Error Reading graph file\n");
        return;
    }

    int source = 0;
    
    fscanf(fp, "%d", &no_of_nodes);
    
    int num_of_blocks = 1;
    int num_of_threads_per_block = no_of_nodes;

    // Make execution parameters according to the number of nodes
    if (no_of_nodes > MAX_THREADS_PER_BLOCK)
    {
        num_of_blocks = (int)ceil(no_of_nodes / (double)MAX_THREADS_PER_BLOCK); 
        num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
    }
        
    // Allocate host memory
    Node* h_graph_nodes = (Node*)malloc(sizeof(Node) * no_of_nodes);
    bool *h_graph_mask = (bool*)malloc(sizeof(bool) * no_of_nodes);
    bool *h_graph_visited = (bool*)malloc(sizeof(bool) * no_of_nodes);
    
    int start, edgeno;   
    // Initialize the memory
    for (unsigned int i = 0; i < no_of_nodes; i++) 
    {
        fscanf(fp, "%d %d", &start, &edgeno);
        h_graph_nodes[i].starting = start;
        h_graph_nodes[i].no_of_edges = edgeno;
        h_graph_mask[i] = false;
        h_graph_visited[i] = false;
    }
    
    // Read the source node from the file
    fscanf(fp, "%d", &source);
    
    // Set the source node as true in the mask
    h_graph_mask[source] = true;
    
    fscanf(fp, "%d", &edge_list_size);
    
    int id, cost;
    int* h_graph_edges = (int*)malloc(sizeof(int) * edge_list_size);
    for (int i = 0; i < edge_list_size; i++)
    {
        fscanf(fp, "%d", &id);
        fscanf(fp, "%d", &cost);
        h_graph_edges[i] = id;
    }
     
    if (fp)
        fclose(fp);    
    
    printf("Read File\n");

    // Copy the Node list to device memory
    Node* d_graph_nodes;
    hipMalloc((void**)&d_graph_nodes, sizeof(Node) * no_of_nodes);
    hipMemcpy(d_graph_nodes, h_graph_nodes, sizeof(Node) * no_of_nodes, hipMemcpyHostToDevice);

    // Copy the Edge List to device Memory
    int* d_graph_edges;
    hipMalloc((void**)&d_graph_edges, sizeof(int) * edge_list_size);
    hipMemcpy(d_graph_edges, h_graph_edges, sizeof(int) * edge_list_size, hipMemcpyHostToDevice);
    
    // Copy the Mask to device memory
    bool* d_graph_mask;
    hipMalloc((void**)&d_graph_mask, sizeof(bool) * no_of_nodes);
    hipMemcpy(d_graph_mask, h_graph_mask, sizeof(bool) * no_of_nodes, hipMemcpyHostToDevice);
    
    // Copy the Visited nodes array to device memory
    bool* d_graph_visited;
    hipMalloc((void**)&d_graph_visited, sizeof(bool) * no_of_nodes);
    hipMemcpy(d_graph_visited, h_graph_visited, sizeof(bool) * no_of_nodes, hipMemcpyHostToDevice);
    
    // Allocate memory for the result on host side
    int* h_cost = (int*)malloc(sizeof(int) * no_of_nodes);
    for (int i = 0; i < no_of_nodes; i++)
        h_cost[i] = -1;
    h_cost[source] = 0;

    // Allocate device memory for result
    int* d_cost;
    hipMalloc((void**)&d_cost, sizeof(int) * no_of_nodes);
    hipMemcpy(d_cost, h_cost, sizeof(int) * no_of_nodes, hipMemcpyHostToDevice);

    // Make a bool to check if the execution is over
    bool *d_over;
    hipMalloc((void**)&d_over, sizeof(bool));

    printf("Copied Everything to GPU memory\n");
    
    // Setup execution parameters
    dim3 grid(num_of_blocks, 1, 1);
    dim3 threads(num_of_threads_per_block, 1, 1);

    // Start the timer
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    float timer_acc = 0.0f;
    int k = 0;
    
    bool stop;
    // Call the Kernel until all the elements of Frontier are not false
    do
    {
        // If no thread changes this value then the loop stops
        stop = false;
        hipMemcpy(d_over, &stop, sizeof(bool), hipMemcpyHostToDevice);
        
        hipEventRecord(start_event);
        Kernel<<<grid, threads>>>(d_graph_nodes, d_graph_edges, d_graph_mask, d_graph_visited, d_cost, d_over, no_of_nodes);
        hipDeviceSynchronize();
        hipEventRecord(stop_event);
        
        hipEventSynchronize(stop_event);
        
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start_event, stop_event);
        timer_acc += elapsed_time;

        // Check if kernel execution generated an error
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(error));
            break;
        }
        
        hipMemcpy(&stop, d_over, sizeof(bool), hipMemcpyDeviceToHost);
        k++;
    }
    while (stop);
    
    printf("Kernel Executed %d times\n", k);

    // Copy result from device to host
    hipMemcpy(h_cost, d_cost, sizeof(int) * no_of_nodes, hipMemcpyDeviceToHost);

    // Stop the timer
    printf("Processing time: %f (ms)\n", timer_acc);
    
    // Store the result into a file
    FILE *fpo = fopen("result.txt", "w");
    for (int i = 0; i < no_of_nodes; i++)
        fprintf(fpo, "%d) cost:%d\n", i, h_cost[i]);
    fclose(fpo);
    printf("Result stored in result.txt\n");
    
    // Cleanup memory
    free(h_graph_nodes);
    free(h_graph_edges);
    free(h_graph_mask);
    free(h_graph_visited);
    free(h_cost);
    hipFree(d_graph_nodes);
    hipFree(d_graph_edges);
    hipFree(d_graph_mask);
    hipFree(d_graph_visited);
    hipFree(d_cost);
    hipFree(d_over);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
}
