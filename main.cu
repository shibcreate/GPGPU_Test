
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000


__global__ void vector_add_cuda(float *out, float *a, float *b, int n) {
    
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(id < N)
    {
        out[id] = a[id] + b[id];
    }
}

int main(){

    // Number of bytes to allocate for N doubles
    size_t bytes = N * sizeof(float);

    float *h_a, *h_b, *out, *h_out, *d_a, *d_b, *d_out; 

    // Allocate data in host pointer
    hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_out, bytes, hipHostMallocDefault);

    // Initialize array
    for(int i = 0; i < N; i++){
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;        
    }

    // Allocate device memory for d_a, d_b, d_out
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_out, bytes);


    // Transfer data from host to device memory
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


    int THREADS = 256;
    int BLOCKS = (N + THREADS - 1) / THREADS;


    // Kernel function
    vector_add_cuda <<< BLOCKS, THREADS >>> (d_out, d_a, d_b, N);


    // Transfer computed data from device to host memory
    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);


    // Check/validate results
    for(int i = 0; i < N; i++)
    {
        if (i < 10)
        {
            printf("h_a[%d] = %.5lf, h_out[%d] = %.5lf\n", i, h_a[i], i, h_out[i]);
        }

        if (out[i] != h_out[i])
        {
            printf(" Error at %d, h_out[%d] = %.5lf, out[%d] = %.5lf\n", i, i, h_out[i], i, out[i]);
            break;
        }

    }

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_out);

    return 0;
}
