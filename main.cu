#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"

int main (int argc, char *argv[]) {
    float *A_h, *B_h, *C_h;   // Host matrices
    float *A_d, *B_d, *C_d;   // Device matrices
    int MatSize;
   
    // Accept matrix size from the user
    if (argc == 1) {
        MatSize = 16;  // Default size
    } else if (argc == 2) {
        MatSize = atoi(argv[1]);  // User-specified size
    } else {
        printf("Usage: ./matMul <Size>\n");
        exit(0);
    }

    int size = MatSize * MatSize * sizeof(float);

    // Allocate memory on the host
    A_h = (float*) malloc(size);
    B_h = (float*) malloc(size);
    C_h = (float*) malloc(size);

    // Initialize the host matrices
    for (int i = 0; i < MatSize * MatSize; i++) {
        A_h[i] = rand() % 100;  // Random values for demo
        B_h[i] = rand() % 100;
    }

    // Allocate memory on the device
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    // Copy data from host to device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);  // 16x16 threads per block
    dim3 blocksPerGrid((MatSize + 15) / 16, (MatSize + 15) / 16);

    // Launch the kernel
    MatMul<<<blocksPerGrid, threadsPerBlock>>>(A_d, B_d, C_d, MatSize);

    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // Print a small portion of the result matrix for verification
    for (int i = 0; i < MatSize && i < 4; i++) {
        for (int j = 0; j < MatSize && j < 4; j++) {
            printf("%f ", C_h[i * MatSize + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    // Free host memory
    free(A_h);
    free(B_h);
    free(C_h);

    return 0;
}
