#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000

__global__ void vector_add_cuda(float *out, float *a, float *b, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (id < n) {
        out[id] = a[id] + b[id];
    }
}

int main() {
    // Number of bytes to allocate for N floats
    size_t bytes = N * sizeof(float);

    float *h_a, *h_b, *h_out, *d_a, *d_b, *d_out;

    // Allocate data in host pointers
    hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_out, bytes, hipHostMallocDefault);

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;        
    }

    // Allocate device memory for d_a, d_b, d_out
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_out, bytes);

    // Transfer data from host to device memory
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int THREADS = 256;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    // Kernel function
    vector_add_cuda <<< BLOCKS, THREADS >>> (d_out, d_a, d_b, N);

    // Transfer computed data from device to host memory
    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);

    // Check/validate results
    for (int i = 0; i < N; i++) {
        if (i < 10) {
            printf("h_a[%d] = %.5f, h_out[%d] = %.5f\n", i, h_a[i], i, h_out[i]);
        }

        // Change 'out[i]' to expected result (1.0 + 2.0)
        if (h_out[i] != 3.0f) {
            printf("Error at %d, h_out[%d] = %.5f, expected = 3.0\n", i, i, h_out[i]);
            break;
        }
    }

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_out);

    return 0;
}
