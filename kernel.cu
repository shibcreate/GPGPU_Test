#include <hip/hip_runtime.h>

#define MAXNUM 9
#define MINNUM 0

// CUDA Kernel for matrix multiplication
__global__ void MatrixMultiplyKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}
