
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void MatMul(const float* A, const float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float value = 0;
        for (int k = 0; k < n; k++) {
            value += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = value;
    }
}
